#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCKSIZE 256

// __device__ void silu_kernel(...);


__global__ void silu_kernel(float *x, float *o, int n) {
    int block_start = blockIdx.x * blockDim.x;
    int thread_id = threadIdx.x;
    int index = block_start + thread_id;

    if (index < n) {
        o[index] = x[index] / (1 + expf(-x[index]));
    }
}

// input and output are allocated on host device. need to be
// copied
void silu(float *input, float *output, int n) {
    // Allocate memory on the device
    float *d_input, *d_output;
    hipMalloc((void**)&d_input, n * sizeof(float));
    hipMalloc((void**)&d_output, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_input, input, n * sizeof(float), hipMemcpyHostToDevice);

    dim3 num_block((n + BLOCKSIZE - 1) / BLOCKSIZE);
    dim3 num_threads(BLOCKSIZE);
    silu_kernel<<<num_block, num_threads>>>(d_input, d_output, n);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
    
    // Copy result back to host
    hipMemcpy(output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}
