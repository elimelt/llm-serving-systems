#include <hip/hip_runtime.h>
#include "silu.h"
#include <iostream>

#define SIZE (8192 * 8192)
#define ITERS 10

int main() {
    // Allocate and initialize host memory
    float* h_input = new float[SIZE];
    for (int i = 0; i < SIZE; i++) {
        h_input[i] = static_cast<float>(i);
    }

    // Allocate device memory and copy input data
    float *d_input, *d_output;
    hipMalloc((void**)&d_input, SIZE * sizeof(float));
    hipMalloc((void**)&d_output, SIZE * sizeof(float));
    hipMemcpy(d_input, h_input, SIZE * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel and measure time across multiple iterations
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float totalTime = 0;
    
    for (int iter = 0; iter < ITERS; iter++) {
        hipEventRecord(start);
        silu(d_input, d_output, SIZE);
        hipEventRecord(stop);


        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        totalTime += milliseconds;
        hipDeviceSynchronize();
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Average time per iteration: %f ms\n", totalTime / ITERS);

    // Free device/host memory
    hipFree(d_input);
    hipFree(d_output);
    delete[] h_input;
    return 0;
}