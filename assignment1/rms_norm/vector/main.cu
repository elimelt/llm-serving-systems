#include<hip/hip_runtime.h>
#include "rms_norm_vector.h"
#include <stdio.h>

#define SIZE 1024 * 1024
#define ITERS 10

int main() {
    float* h_input = new float[SIZE];
    float* h_weight = new float[SIZE];

    for (int i = 0; i < SIZE; i++) {
        h_input[i] = static_cast<float>(i + 1);
        h_weight[i] = 1.0f;
    }

    size_t matrixSize = SIZE * sizeof(float);
    float *d_input, *d_weight, *d_output;
    hipMalloc((void**)&d_input, matrixSize);
    hipMalloc((void**)&d_weight, matrixSize);
    hipMalloc((void**)&d_output, matrixSize);

    hipMemcpy(d_input, h_input, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_weight, h_weight, matrixSize, hipMemcpyHostToDevice);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float totalTime = 0;

    for (int iter = 0; iter < ITERS; iter++) {
        hipEventRecord(start);
        rms_norm_vector(d_input, d_weight, d_output, SIZE, 0.000001f);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        totalTime += milliseconds;
        hipDeviceSynchronize();
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Average time per iteration: %f ms\n", totalTime / ITERS);

    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
    delete[] h_input;
    delete[] h_weight;
    return 0;
}