#include "hip/hip_runtime.h"
#include "rms_norm_vector.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define ELEMENT_PER_BLOCK 256

__global__ void rms_norm_vector_kernel(float *input, float *weight, float *output, int cols, float epsilon) {
    __shared__ float sdata[ELEMENT_PER_BLOCK];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = (i < cols) ? input[i] * input[i] : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // apply RMS normalization
    float rms = sqrtf(sdata[0] / cols + epsilon);
    output[i] = (i < cols) ? input[i] * weight[i] / rms : 0;
}

void rms_norm_vector(float *input, float *weight, float *output, int cols, float epsilon) {
    // Launch kernel
    dim3 grid((cols + ELEMENT_PER_BLOCK - 1) / ELEMENT_PER_BLOCK);
    dim3 block(ELEMENT_PER_BLOCK);
    rms_norm_vector_kernel<<<grid, block>>>(input, weight, output, cols, epsilon);

    // Check for errors in kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
}