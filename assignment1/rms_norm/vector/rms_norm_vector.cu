#include "hip/hip_runtime.h"
#include "rms_norm_vector.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define ELEMENT_PER_BLOCK 256

__global__ void rms_norm_vector_kernel(float *input, float *weight, float *output, int cols, float epsilon) {
    __shared__ float sdata[ELEMENT_PER_BLOCK];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = (i < cols) ? input[i] * input[i] : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // apply RMS normalization
    float rms = sqrtf(sdata[0] / cols + epsilon);
    output[i] = (i < cols) ? input[i] * weight[i] / rms : 0;
}

void rms_norm_vector(float *input, float *weight, float *output, int cols, float epsilon) {
    // Allocate device memory
    size_t vectorSize = cols * sizeof(float);
    float *d_input, *d_weight, *d_output;
    hipMalloc((void**)&d_input, vectorSize);
    hipMalloc((void**)&d_weight, vectorSize);
    hipMalloc((void**)&d_output, vectorSize);
    
    // Copy data from host to device
    hipMemcpy(d_input, input, vectorSize, hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, vectorSize, hipMemcpyHostToDevice);
    
    // Launch kernel
    dim3 grid((cols + ELEMENT_PER_BLOCK - 1) / ELEMENT_PER_BLOCK);
    dim3 block(ELEMENT_PER_BLOCK);
    rms_norm_vector_kernel<<<grid, block>>>(d_input, d_weight, d_output, cols, epsilon);
    
    // Wait for kernel completion and check for errors.
    hipDeviceSynchronize();
    
    // Check for errors in kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
    
    // Copy the result back to host
    hipMemcpy(output, d_output, vectorSize, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
}