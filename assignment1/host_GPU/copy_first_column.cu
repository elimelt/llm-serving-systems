#include "copy_first_column.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>

void copy_first_column(float *h_A, float *d_A, int rows, int cols)
{
  hipMalloc((void **)&d_A, rows * cols * sizeof(float));
  hipMemcpy(d_A, h_A, rows * cols * sizeof(float), hipMemcpyHostToDevice);

  float *h_first_column = (float *)malloc(rows * sizeof(float));

  for (int i = 0; i < rows; i++)
  {
    hipMemcpy(&h_first_column[i], &d_A[i * cols], sizeof(float), hipMemcpyDeviceToHost);
  }

  hipFree(d_A);

  free(h_first_column);
}