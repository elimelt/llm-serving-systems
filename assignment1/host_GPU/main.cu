#include "hip/hip_runtime.h"
#include "copy_first_column.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define ROWS 8192 
#define COLS 65536
#define SIZE (ROWS * COLS)

int main() {
    float *output, *input;
    
    // Allocate pinned memory for input with default flags
    hipError_t err = hipHostAlloc((void**)&input, SIZE * sizeof(float), hipHostMallocDefault);
    if (err != hipSuccess) {
        std::cerr << "Error allocating pinned host memory: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    
    // Allocate device memory for output
    err = hipMalloc((void**)&output, ROWS * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Error allocating device memory: " << hipGetErrorString(err) << std::endl;
        hipHostFree(input);
        return 1;
    }

    // Initialize input data
    for (int i = 0; i < SIZE; i++) {
        input[i] = static_cast<float>(i + 1);
    }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Warmup run
    copy_first_column(input, output, ROWS, COLS);
    
    // Record start event
    hipEventRecord(start);
    
    // Execute the copy operation
    copy_first_column(input, output, ROWS, COLS);
    
    // Record stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Calculate and print execution time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Execution time: " << milliseconds * 1000.0f << " microseconds" << std::endl;

    // Verify the results
    float *h_output = new float[ROWS];
    err = hipMemcpy(h_output, output, ROWS * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Error copying from device to host: " << hipGetErrorString(err) << std::endl;
    } else {
        // Verify first few elements
        for (int i = 0; i < std::min(10, ROWS); i++) {
            float expected = input[i * COLS];
            if (h_output[i] != expected) {
                std::cerr << "Verification failed at index " << i 
                          << ": expected " << expected 
                          << ", got " << h_output[i] << std::endl;
                break;
            }
        }
    }

    // Cleanup
    delete[] h_output;
    hipHostFree(input);
    hipFree(output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}